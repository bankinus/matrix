
#include <hip/hip_runtime.h>
#include <iostream>

__global__
void square(float *d_in, float *d_out)
{
    d_out[threadIdx.x] = d_in[threadIdx.x] * d_in[threadIdx.x];
}

int main(int argc, char *argv[])
{
    const int arraysize = 64;
    const int bytesize = sizeof(float) * arraysize;
    float h_in[64];
    float h_out[64];
    for (int i = 0; i<bytesize; i++){
        h_in[i] = i;
    }
    float *d_in;
    float *d_out;

    hipMalloc((void **) &d_in, bytesize);
    hipMalloc((void **) &d_out, bytesize);
    hipMemcpy(d_in, h_in, bytesize, hipMemcpyHostToDevice);

    square<<< 1, arraysize >>>(d_in, d_out);

    hipMemcpy(h_out, d_out, bytesize, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    for (int i = 0; i < arraysize; i += 4){
        std::cout << h_out[i] << "\t" << h_out[i+1] << "\t" << h_out[i+2] << "\t" << h_out[i+3] << std::endl;
    }

    return 0;
}

